#include "hip/hip_runtime.h"
#include "myCuda.h"

// create phase mask
__global__ void phase_mask(double *uf, double *vf, double *wf, int *phase, 
  int N3)
{
  int pp = threadIdx.x + blockIdx.x*blockDim.x;;

  if (pp < N3) {
    phase[pp] = (phase[pp] == -1);
  }
}
