#include "hip/hip_runtime.h"
#include "myCuda.h"
//#include "time.h"

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

extern "C"
void cuda_dev_malloc(void)
{
  // allocate device memory on device
  hipSetDevice(dev_start);

  hipMalloc((void**) &_parts, sizeof(part_struct) * nparts);
  hipMalloc((void**) &_dom, sizeof(dom_struct));
  hipMalloc((void**) &_uf, sizeof(double) * dom.Gcc.s3);
  hipMalloc((void**) &_vf, sizeof(double) * dom.Gcc.s3);
  hipMalloc((void**) &_wf, sizeof(double) * dom.Gcc.s3);
  hipMalloc((void**) &_phase, sizeof(int) * dom.Gcc.s3);

}

extern "C"
void cuda_dom_push(void)
{
  hipSetDevice(dev_start);
  // copy host data to device
  hipMemcpy(_dom, &dom, sizeof(dom_struct), hipMemcpyHostToDevice);
}

extern "C"
void cuda_flow_push(void)
{
  hipSetDevice(dev_start);

  hipMemcpy(_uf, uf, sizeof(double) * dom.Gcc.s3, hipMemcpyHostToDevice);
  hipMemcpy(_vf, vf, sizeof(double) * dom.Gcc.s3, hipMemcpyHostToDevice);
  hipMemcpy(_wf, wf, sizeof(double) * dom.Gcc.s3, hipMemcpyHostToDevice);
  hipMemcpy(_phase, phase, sizeof(int) *dom.Gcc.s3, hipMemcpyHostToDevice);
}

extern "C"
void cuda_flow_pull(void)
{
  hipMemcpy(uf, _uf, sizeof(double) * dom.Gcc.s3, hipMemcpyDeviceToHost);
  hipMemcpy(vf, _vf, sizeof(double) * dom.Gcc.s3, hipMemcpyDeviceToHost);
  hipMemcpy(wf, _wf, sizeof(double) * dom.Gcc.s3, hipMemcpyDeviceToHost);
  hipMemcpy(phase, _phase, sizeof(int) *dom.Gcc.s3, hipMemcpyDeviceToHost);
}

extern "C"
void cuda_part_push(void)
{
  hipSetDevice(dev_start);
  hipMemcpy(_parts, parts, sizeof(part_struct) * nparts, 
    hipMemcpyHostToDevice);
}

extern "C"
void cuda_part_pull(void)
{
  hipMemcpy(parts, _parts, sizeof(part_struct) * nparts, 
    hipMemcpyDeviceToHost);
}

void cuda_phase_averaged_vel(void)
{
  // Paralleize over flow nodes
  int threads = MAX_THREADS_1D;
  int blocks = (int) ceil((double) dom.Gcc.s3 / (double) threads);
  if (threads > dom.Gcc.s3) {
    threads = dom.Gcc.s3;
    blocks = 1;
  }
  dim3 numBlocks(blocks);
  dim3 dimBlocks(threads);

  // create phase mask
  phase_mask<<<numBlocks, dimBlocks>>>(_uf, _vf, _wf, _phase, dom.Gcc.s3);

}

extern "C"
void cuda_dev_free(void)
{
  hipFree(_dom);
  hipFree(_uf);
  hipFree(_vf);
  hipFree(_wf);
  hipFree(_phase);
  hipFree(_parts);

  hipDeviceReset();
}
